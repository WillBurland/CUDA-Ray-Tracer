#include "hip/hip_runtime.h"
#include "renderer.cuh"

#include "hit_record.cuh"

__device__ bool lambertianScatter(Ray ray, HitRecord* hitRecord, float3* attenuation, Ray* scattered, ulong* seed) {
	float3 scatterDir = hitRecord->normal + randUnitVec(seed);
	if (nearZero(scatterDir))
		scatterDir = hitRecord->normal;

	scattered->origin = hitRecord->p;
	scattered->direction = scatterDir;
	scattered->invDirection = inv(scatterDir);
	*attenuation = hitRecord->material.albedo;
	return true;
}

__device__ bool metalScatter(Ray ray, HitRecord* hitRecord, float3* attenuation, Ray* scattered, ulong* seed) {
	float3 reflected = reflect(unit(ray.direction), hitRecord->normal);
	scattered->origin = hitRecord->p;
	scattered->direction = hitRecord->material.fuzz > 0.0f ? reflected + randVecInUnitSphere(seed) * hitRecord->material.fuzz : reflected;
	scattered->invDirection = inv(scattered->direction);
	*attenuation = hitRecord->material.albedo;
	return dot(scattered->direction, hitRecord->normal) > 0;
}

__device__ bool transparentScatter(Ray ray, HitRecord* hitRecord, float3* attenuation, Ray* scattered, ulong* seed) {
	float refractionRatio = hitRecord->frontFace ? (1.0f / hitRecord->material.ior) : hitRecord->material.ior;

	float3 unitDirection = unit(ray.direction);
	float cosTheta = fminf(dot(unitDirection * -1, hitRecord->normal), 1.0f);
	float sinTheta = sqrtf(1.0f - cosTheta * cosTheta);

	bool cannotRefract = refractionRatio * sinTheta > 1.0f;
	float3 direction;

	if (cannotRefract || reflectance(cosTheta, refractionRatio) > randFloat(seed)) {
		direction = reflect(unitDirection, hitRecord->normal);
	} else {
		direction = refract(unitDirection, hitRecord->normal, refractionRatio);
	}

	scattered->origin = hitRecord->p;
	scattered->direction = direction;
	scattered->invDirection = inv(scattered->direction);
	*attenuation = make_float3(1.0f);
	return true;
}

__device__ float3 rayColour(Ray ray, Scene* scene, ulong* seed) {
	float3 unitDirection = unit(ray.direction);
	float3 rayColour = make_float3(1.0f);

	int currentBounces = 0;
	HitRecord hitRecord;

	while (currentBounces < MAX_BOUNCES) {
		if (hitAnything(&hitRecord, ray, 0.001f, INFINITY, scene)) {
			Ray scattered(ray.origin, ray.direction, ray.invDirection);
			float3 attenuation;
			switch (hitRecord.material.type) {
				case LAMBERTIAN: {
					if (lambertianScatter(ray, &hitRecord, &attenuation, &scattered, seed)) {
						ray = scattered;
						rayColour *= attenuation;
						currentBounces++;
						continue;
					}
					return make_float3(0.0f);
				}
				case METAL: {
					if (metalScatter(ray, &hitRecord, &attenuation, &scattered, seed)) {
						ray = scattered;
						rayColour *= attenuation;
						currentBounces++;
						continue;
					}
					return make_float3(0.0f);
				}
				case TRANSPARENT: {
					if (transparentScatter(ray, &hitRecord, &attenuation, &scattered, seed)) {
						ray = scattered;
						rayColour *= attenuation;
						currentBounces++;
						continue;
					}
					return hitRecord.material.albedo;
				}
				case EMISSIVE: {
					return rayColour + hitRecord.material.albedo;
				}
			}
			continue;
		}
		break;
	}

	float2 uvCoords = uv(ray.direction);
	float4 texColour = tex2D<float4>(scene->hdrTex, uvCoords.x, uvCoords.y);
	float3 hdriColour = make_float3(texColour.x, texColour.y, texColour.z);
	
	return rayColour * hdriColour;
}

__global__ void shadePixel(unsigned char* image, Scene* scene) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= IMAGE_WIDTH || y >= IMAGE_HEIGHT)
		return;

	int idx = (IMAGE_HEIGHT - 1 - y) * IMAGE_WIDTH + x;
	ulong seed = nextSeed((ulong)(idx * idx));

	float3 pixelColour = make_float3(0.0f);
	float3 colourToAdd = make_float3(0.0f);

	for (int i = 0; i < SAMPLES_PER_PIXEL; i++) {
		float u = ((float)x + randFloat(&seed)) / IMAGE_WIDTH;
		float v = ((float)y + randFloat(&seed)) / IMAGE_HEIGHT;

		Ray ray = Ray(scene->camera, u, v, &seed);
		colourToAdd = rayColour(ray, scene, &seed);

		if (isnan(colourToAdd.x)) colourToAdd.x = 0.0f;
		if (isnan(colourToAdd.y)) colourToAdd.y = 0.0f;
		if (isnan(colourToAdd.z)) colourToAdd.z = 0.0f;

		pixelColour += colourToAdd;
	}

	pixelColour /= (float)SAMPLES_PER_PIXEL;

	pixelColour.x = sqrtf(pixelColour.x);
	pixelColour.y = sqrtf(pixelColour.y);
	pixelColour.z = sqrtf(pixelColour.z);

	pixelColour = clamp(pixelColour, 0.0f, 1.0f);

	image[idx * 3 + 0] = pixelColour.x * 255.0f;
	image[idx * 3 + 1] = pixelColour.y * 255.0f;
	image[idx * 3 + 2] = pixelColour.z * 255.0f;
}

void renderImage(unsigned char* image, Scene* scene) {
	dim3 block(16, 16);
	dim3 grid((IMAGE_WIDTH + block.x - 1) / block.x,
			  (IMAGE_HEIGHT + block.y - 1) / block.y);

	shadePixel<<<grid, block>>>(image, scene);
	hipDeviceSynchronize();
}
